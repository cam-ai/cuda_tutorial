#include "hip/hip_runtime.h"
//https://stackoverflow.com/questions/69189880/what-is-the-difference-between-mapped-memory-and-managed-memory
#include <iostream>
#include <cassert>

__global__
void kernel(char* __restrict__ data, int pagesize, int numpages){
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < numpages){
        data[tid * pagesize] += 1;
    }
}

int main(){
    const int pagesize = 4096;
    const int numpages = 1024 * 64;
    const int bytes = pagesize * numpages;
    hipError_t status = hipSuccess;
    float elapsed = 0.0f;
    const int iterations = 5;

    char* devicedata; 
    status = hipMalloc(&devicedata, bytes);
    assert(status == hipSuccess);

    char* pinneddata; 
    status = hipHostMalloc(&pinneddata, bytes);
    assert(status == hipSuccess);

    char* manageddata;
    status = hipMallocManaged(&manageddata, bytes);
    assert(status == hipSuccess);

    status = hipMemPrefetchAsync(manageddata, bytes, hipCpuDeviceId);
    //status = hipMemPrefetchAsync(manageddata, bytes, 0);
    assert(status == hipSuccess);

    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    for(int iteration = 0; iteration < iterations; iteration++){
        hipEventRecord(event1);
        kernel<<<numpages / 256, 256>>>(pinneddata, pagesize, numpages);
        hipEventRecord(event2);
        status = hipEventSynchronize(event2);
        assert(status == hipSuccess);
        hipEventElapsedTime(&elapsed, event1, event2);
        
        float bandwith = (numpages / elapsed) * 1000.0f / 1024.f / 1024.f;
        std::cerr << "pinned: " << elapsed << ", throughput " << bandwith << " GB/s" << "\n";
    }

    for(int iteration = 0; iteration < iterations; iteration++){
        hipEventRecord(event1);
        kernel<<<numpages / 256, 256>>>(manageddata, pagesize, numpages);
        hipEventRecord(event2);
        status = hipEventSynchronize(event2);
        assert(status == hipSuccess);
        hipEventElapsedTime(&elapsed, event1, event2);

        float bandwith = (numpages / elapsed) * 1000.0f / 1024.f / 1024.f;
        std::cerr << "managed: " << elapsed << ", throughput " << bandwith << " MB/s" << "\n";

        //status = hipMemPrefetchAsync(manageddata, bytes, hipCpuDeviceId);
        assert(status == hipSuccess);     
    }

    for(int iteration = 0; iteration < iterations; iteration++){
        hipEventRecord(event1);
        kernel<<<numpages / 256, 256>>>(devicedata, pagesize, numpages);
        hipEventRecord(event2);
        status = hipEventSynchronize(event2);
        assert(status == hipSuccess);
        hipEventElapsedTime(&elapsed, event1, event2);
        
        float bandwith = (numpages / elapsed) * 1000.0f / 1024.f / 1024.f;
        std::cerr << "device: " << elapsed << ", throughput " << bandwith << " MB/s" << "\n";
    }

    hipHostFree(pinneddata);
    hipFree(manageddata);
    hipFree(devicedata);
    hipEventDestroy(event1);
    hipEventDestroy(event2);

}
