//https://stackoverflow.com/questions/69189880/what-is-the-difference-between-mapped-memory-and-managed-memory

#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

__device__ float factor = 0.0;


__global__ void globalMemory()
{
    printf("device global memory : %.2f\n", factor);
    factor += 1.2;
}


int main()
{
    dim3 block(1,1);
    dim3 grid(1,1);
    float h_factor = 3.6;

    hipSetDevice(1);

    //注意第一个参数并没有 & 取地址 它是一个符号，只需要填入变量名字即可
    hipMemcpyToSymbol(HIP_SYMBOL(factor), &h_factor, sizeof(float), 0, hipMemcpyHostToDevice);

    globalMemory<<<grid, block>>>();

    hipDeviceSynchronize();

    hipMemcpyFromSymbol(&h_factor, HIP_SYMBOL(factor), sizeof(float), 0, hipMemcpyDeviceToHost);
    printf("host factor is %.2f\n", h_factor);

    //get global address
    float *pDeviceFactor;
    hipGetSymbolAddress((void**)&pDeviceFactor, HIP_SYMBOL(factor));
    hipMemcpy(&h_factor, pDeviceFactor, sizeof(float), hipMemcpyDeviceToHost);
    printf("host factor is %.2f\n", h_factor);

    //pointer attribute
    hipPointerAttribute_t ptAttr;
    hipPointerGetAttributes(&ptAttr, pDeviceFactor);
    printf("pointer attribute:device=%d, devicePointer=%p, type= %d\n",
        ptAttr.device, ptAttr.devicePointer, ptAttr.type);

    hipDeviceReset();
    return 0;
}
