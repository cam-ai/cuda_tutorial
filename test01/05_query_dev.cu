
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) 
{
	printf("successfull %d \n", threadIdx.x + blockDim.x* blockIdx.x);
	printf("blockDim = [%d %d]\n", blockDim.x, blockDim.y);
}

int main(void)
{
	printf("Hello world from CPU\n");

	helloFromGPU<<<2, 10>>>(); // <<<number of blocks, number of threads per block>>>
				   // <<<blockIdx, threadIdx.x>>>

	hipDeviceSynchronize();// cpu gpu同步
	hipDeviceReset();
	return 0;
}
