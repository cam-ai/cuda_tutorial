
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void) //__global__ 关键字
{
	printf("Hello world from GPU\n");
}

int main(void)
{
	printf("Hello world from CPU\n");

	// 内核调用
	helloFromGPU<<<2, 10>>>(); // <<<number of blocks, number of threads per block(1024 the biggest )>>>
				   // <<<blockIdx, threadIdx.x>>>
				   // <<<grid, block>>

	hipDeviceSynchronize();// cpu gpu同步
	// 释放所有与当前进程相关的GPU资源
	hipDeviceReset();
	return 0;
}
